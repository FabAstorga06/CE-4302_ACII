#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10 // Tamaño de la matriz

/************************************************************************/

// Imprime las matrices
void print_mat(float mat[N][N] ) {
    for(int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            printf ("%f \t", mat[i][j]);
        }
        printf ("\n");
    }
    printf ("\n\n");    
}

// Suma matrices
void cpu_add(float A[N][N], float B[N][N], float C[N][N] ) {
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            C[i][j] = A[i][j] + B[i][j];
        }
    }
}

// Kernel
__global__ void add_mtx(float A[N][N], float B[N][N], float C[N][N]) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

/************************************************************************/

int main() {

    float A [N][N], B [N][N], C [N][N]; 
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            A [i][j] = i * N + j;
            B [i][j] = j * N + i;
        }
    }

    clock_t start_h = clock();
    cpu_add (A, B, C);
    clock_t end_h = clock();

    clock_t start_d = clock();
    // Kernel
    dim3 threads_block(16, 16); // Hilos y bloques
    dim3 num_blocks(N / threads_block.x, N / threads_block.y);
    add_mtx<<<num_blocks, threads_block>>>(A, B, C);
    clock_t end_d = clock();

    //Comparacion de tiempos 
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
    printf("Tiempo de GPU = %fs \t Tiempo de CPU = %fs\n", time_d, time_h);

}
