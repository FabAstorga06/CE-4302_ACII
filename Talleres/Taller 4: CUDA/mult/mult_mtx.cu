#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 4
int *A, *B, *C, *D;	

/************************************************************************/

// Inicializa matrices
void init_mtx() {
	A = (int *) malloc((SIZE*SIZE) * sizeof(int));	
	B = (int *) malloc((SIZE*SIZE) * sizeof(int));
	C = (int *) malloc((SIZE*SIZE) * sizeof(int));
	D = (int *) malloc((SIZE*SIZE) * sizeof(int));

	for (int i = 0; i < (n*n); ++i ) {
		A[i] = i;
		B[i] = i;
		C[i] = 0;
	}
}

// Kernel que multiplica matrices
__global__ void mult_mtx(int *D, int *A, int *B, int *C ) {
	int value = 0;
	int rows = blockIdx.y * blockDim.y + threadIdx.y;
	int cols = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < SIZE; ++i) {
		value += A[(SIZE+i)*rows] * B[(SIZE*i)+cols];
	}	
	D[rows*(SIZE+cols)] = value + C[(rows*SIZE)+cols];	

/************************************************************************/

int main(int argc, char* argv[])
{

	init_mtx();

	int *d_A, *d_B, *d_C, *d_D;
	size_t size = (SIZE*SIZE) * sizeof(int);

	hipMalloc(&d_A, size);		
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	

	hipMalloc(&d_B, size);		
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc(&d_C, size);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

	size = (SIZE*SIZE) * sizeof(int);
	hipMalloc(&d_D, size);		

	//kernel 
	dim3 dim_block(SIZE, SIZE);	
	dim3 dim_grid(1, 1);	
	mult_mtx<<<dim_grid, dim_block>>>(d_D, d_A, d_B, d_C);

	hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);

	//Libera memoria
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
	return 0;
}
